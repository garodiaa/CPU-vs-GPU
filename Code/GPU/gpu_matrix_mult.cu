#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void gpu_matrix_mult(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    float *A, *B, *C;

    // Allocate Unified Memory (accessible from CPU/GPU)
    hipMallocManaged(&A, N * N * sizeof(float));
    hipMallocManaged(&B, N * N * sizeof(float));
    hipMallocManaged(&C, N * N * sizeof(float));

    // Initialize matrices with random values
    for (int i = 0; i < N * N; i++) {
        A[i] = (float)rand() / RAND_MAX;
        B[i] = (float)rand() / RAND_MAX;
    }

    // Configure kernel dimensions (16x16 threads per block)
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    gpu_matrix_mult<<<grid, block>>>(A, B, C, N);
    hipDeviceSynchronize();  // Wait for GPU to finish

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);

    printf("GPU Time: %.3f ms\n", time);

    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}